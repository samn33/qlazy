#include "hip/hip_runtime.h"
/*
 *  qstate_gpu.cu
 */
#pragma GCC diagnostic ignored "-Wunused-but-set-variable"

#ifdef __cplusplus
extern "C" {
#endif

#include "qlazy.h"

#define BLOCKSIZE 32

__constant__ hipDoubleComplex d_U[16];
__constant__ int d_qid[MAX_QUBIT_NUM];
__constant__ int d_cid[MAX_QUBIT_NUM];

__global__ void cuda_qstate_operate_unitary2(hipDoubleComplex* d_camp_out, hipDoubleComplex* d_camp_in,
					     int qubit_num, int state_num, int n)
{
  int		nn, i, p, pp, sign, off;
  hipDoubleComplex	camp;

  nn = qubit_num - n - 1;
  
  i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < state_num) {
    
    p = (i >> nn) % 2;
    pp = p ^ 1;
    sign = (pp << 1) - 1; // pp=0 -> -1, pp=1 -> +1
    off = sign * (1 << nn);

    camp = hipCmul(d_U[IDX2(p,p)], d_camp_in[i]);
    d_camp_out[i] = hipCadd(camp, hipCmul(d_U[IDX2(p,pp)], d_camp_in[i + off]));
  }
}

__global__ void cuda_qstate_operate_unitary4(hipDoubleComplex* d_camp_out, hipDoubleComplex* d_camp_in,
					     int qubit_num, int state_num, int m, int n)
{
  int			mm, nn, i, p, pp, q, qq;
  int			l, sign_p, sign_q, off_p, off_q;
  hipDoubleComplex	camp;
  
  mm = qubit_num - m - 1;
  nn = qubit_num - n - 1;

  i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < state_num) {

    p = (i >> mm) % 2;
    pp = p ^ 1;
    q = (i >> nn) % 2;
    qq = q ^ 1;

    l = (p << 1) + q;

    sign_p = (pp << 1) - 1;
    sign_q = (qq << 1) - 1;

    off_p = sign_p * (1 << mm);
    off_q = sign_q * (1 << nn);

    camp = hipCmul(d_U[IDX4(l, l)], d_camp_in[i]);
    camp = hipCadd(camp, hipCmul(d_U[IDX4(l, (l^1))], d_camp_in[i + off_q]));
    camp = hipCadd(camp, hipCmul(d_U[IDX4(l, (l^2))], d_camp_in[i + off_p]));
    d_camp_out[i] = hipCadd(camp, hipCmul(d_U[IDX4(l, (l^3))], d_camp_in[i + off_q + off_p]));
  }
}

__global__ void cuda_qstate_operate_controlled_gate(hipDoubleComplex* d_camp_out, hipDoubleComplex* d_camp_in,
						    int qubit_num, int state_num, int m, int n)
{
  int			mm, nn, i, p, q, qq;
  int			l, sign_q, off_q;
  hipDoubleComplex	camp;
  
  mm = qubit_num - m - 1;
  nn = qubit_num - n - 1;

  i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < state_num) {

    p = (i >> mm) % 2;
    q = (i >> nn) % 2;
    qq = q ^ 1;

    l = (p << 1) + q;

    sign_q = (qq << 1) - 1;
    off_q = sign_q * (1 << nn);

    camp = hipCmul(d_U[IDX4(l, l)], d_camp_in[i]);
    d_camp_out[i] = hipCadd(camp, hipCmul(d_U[IDX4(l, (l^1))], d_camp_in[i + off_q]));
  }
}

__global__ void cuda_qstate_update_prob_array(hipDoubleComplex* d_camp, double* d_prob_array,
					      int state_num)
{
  double p = 0.0;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < state_num) {
    p = hipCabs(d_camp[i]);
    d_prob_array[i] = p * p;
  }
}

__global__ void cuda_qstate_operate_measure(hipDoubleComplex* d_camp, double* d_prob_array,
					    char* d_mchar_shots, float* d_rand, int qubit_num,
					    int state_num, int cmem_num, int mnum, int shots)
{
  int		i, j, k;
  double        r = 0.0;
  int           idx, up;

  i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < shots) {

    r = (double)d_rand[i];
    idx = 0;
    for (j=0; j<qubit_num; j++) {
      up = 1 << (qubit_num - 1 - j);
      if (r >= d_prob_array[idx + up]) {
	idx = idx + up;
      }
    }
    
    for (k=0; k<mnum; k++) {
      d_mchar_shots[i * cmem_num + d_cid[k]] = (idx >> (qubit_num - d_qid[k] - 1)) % 2;
    }
  }
}

static bool _qstate_update_prob_array_gpu(QState* qstate)
{
  hipDoubleComplex*	d_buffer_0   = qstate->d_buffer_0;
  hipDoubleComplex*	d_buffer_1   = qstate->d_buffer_1;
  hipDoubleComplex*      d_camp	     = NULL;
  double*		d_prob_array = qstate->d_prob_array;
  int			blocksize    = BLOCKSIZE;
  dim3			block (blocksize, 1, 1);
  dim3			grid ((qstate->state_num + block.x - 1) / block.x, 1, 1);
  int                   i;
  double                prob_pre, prob_now;

  if (qstate->d_prob_updated == false) {
    if (qstate->d_buf_id == 0) d_camp = d_buffer_0;
    else d_camp = d_buffer_1;
  
    cuda_qstate_update_prob_array<<< grid, block >>>(d_camp, d_prob_array,
  						     qstate->state_num);
    checkCudaErrors(hipMemcpy(qstate->prob_array, qstate->d_prob_array, sizeof(double) * qstate->state_num,
  			       hipMemcpyDeviceToHost));
  
    prob_pre = qstate->prob_array[0];
    qstate->prob_array[0] = 0.0;
    for (i=1; i<qstate->state_num; i++) {
      prob_now = qstate->prob_array[i];
      qstate->prob_array[i] = qstate->prob_array[i - 1] + prob_pre;
      prob_pre = prob_now;
    }
  
    checkCudaErrors(hipMemcpy(qstate->d_prob_array, qstate->prob_array, sizeof(double) * qstate->state_num,
  			       hipMemcpyHostToDevice));
    
    qstate->d_prob_updated = true;
  }

  SUC_RETURN(true);
}

static bool _qstate_operate_unitary_gpu_static(QState* qstate, int dim, int m, int n)
{
  int			qubit_num  = qstate->qubit_num;
  int			state_num  = qstate->state_num;
  hipDoubleComplex*	d_buffer_0 = qstate->d_buffer_0;
  hipDoubleComplex*	d_buffer_1 = qstate->d_buffer_1;
  int			blocksize  = BLOCKSIZE;
  dim3			block (blocksize, 1, 1);
  dim3			grid ((state_num + block.x - 1) / block.x, 1, 1);

  if ((qstate == NULL) || (qstate->use_gpu == false) || (dim < 0))
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  if (dim == 2) {
    /* 0 -> 1 */
    if (qstate->d_buf_id == 0) {
      cuda_qstate_operate_unitary2<<< grid, block >>>(d_buffer_1, d_buffer_0,
      						      qubit_num, state_num, m);
      qstate->d_buf_id = 1;
      qstate->d_camp = qstate->d_buffer_1;
    }
    /* 1 -> 0 */
    else {
      cuda_qstate_operate_unitary2<<< grid, block >>>(d_buffer_0, d_buffer_1,
						      qubit_num, state_num, m);
      qstate->d_buf_id = 0;
      qstate->d_camp = qstate->d_buffer_0;
    }
  }
  else if (dim == 4) {
    /* 0 -> 1 */
    if (qstate->d_buf_id == 0) {
      cuda_qstate_operate_unitary4<<< grid, block >>>(d_buffer_1, d_buffer_0,
      						      qubit_num, state_num, m, n);
      qstate->d_buf_id = 1;
      qstate->d_camp = qstate->d_buffer_1;
    }
    /* 1 -> 0 */
    else {
      cuda_qstate_operate_unitary4<<< grid, block >>>(d_buffer_0, d_buffer_1,
						      qubit_num, state_num, m, n);
      qstate->d_buf_id = 0;
      qstate->d_camp = qstate->d_buffer_0;
    }
  }
  else {
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);
  }

  qstate->d_prob_updated = false;

  SUC_RETURN(true);
}

static bool _qstate_operate_controlled_gate_gpu_static(QState* qstate, int m, int n)
{
  int			qubit_num  = qstate->qubit_num;
  int			state_num  = qstate->state_num;
  hipDoubleComplex*	d_buffer_0 = qstate->d_buffer_0;
  hipDoubleComplex*	d_buffer_1 = qstate->d_buffer_1;
  int			blocksize  = BLOCKSIZE;
  dim3			block (blocksize, 1, 1);
  dim3			grid ((state_num + block.x - 1) / block.x, 1, 1);

  if ((qstate == NULL) || (qstate->use_gpu == false))
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  /* 0 -> 1 */
  if (qstate->d_buf_id == 0) {
    cuda_qstate_operate_controlled_gate<<< grid, block >>>(d_buffer_1, d_buffer_0,
							   qubit_num, state_num, m, n);
    qstate->d_buf_id = 1;
    qstate->d_camp = qstate->d_buffer_1;
  }
  /* 1 -> 0 */
  else {
    cuda_qstate_operate_controlled_gate<<< grid, block >>>(d_buffer_0, d_buffer_1,
							   qubit_num, state_num, m, n);
    qstate->d_buf_id = 0;
    qstate->d_camp = qstate->d_buffer_0;
  }

  qstate->d_prob_updated = false;

  SUC_RETURN(true);
}

bool qstate_operate_controlled_gate_gpu(QState* qstate, COMPLEX* U, int m, int n)
{
  int			i;
  hipDoubleComplex*	h_U = NULL;

  if ((qstate == NULL) || (qstate->use_gpu == false) || (U == NULL))
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  checkCudaErrors(hipHostMalloc((void**)&h_U, sizeof(hipDoubleComplex) * 16));

  for (i=0; i<16; i++) {
    h_U[i] = make_hipDoubleComplex(creal(U[i]), cimag(U[i]));
  }
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_U), h_U, sizeof(hipDoubleComplex) * 16));

  _qstate_operate_controlled_gate_gpu_static(qstate, m, n);

  checkCudaErrors(hipHostFree(h_U));

  SUC_RETURN(true);
}

bool qstate_operate_unitary_gpu(QState* qstate, COMPLEX* U, int dim, int m, int n)
{
  int			i;
  hipDoubleComplex*	h_U = NULL;

  if ((qstate == NULL) || (qstate->use_gpu == false) || (U == NULL) || (dim < 0))
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  checkCudaErrors(hipHostMalloc((void**)&h_U, sizeof(hipDoubleComplex) * 16));

  for (i=0; i<dim*dim; i++) {
    h_U[i] = make_hipDoubleComplex(creal(U[i]), cimag(U[i]));
  }
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_U), h_U, sizeof(hipDoubleComplex) * dim * dim));

  _qstate_operate_unitary_gpu_static(qstate, dim, m, n);

  checkCudaErrors(hipHostFree(h_U));

  SUC_RETURN(true);
}

static bool _qstate_get_measured_char_gpu(QState* qstate, int mnum, int* qid, char* mchar)
/* not update qstate, get measured char only */
{
  double	r = 0.0;
  int           idx, up;
  int		i;

  if (qstate == NULL) ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  if (qstate->d_prob_updated == false) {
    _qstate_update_prob_array_gpu(qstate);
  }

  r = genrand_real1();
  idx = 0;
  for (i=0; i<qstate->qubit_num; i++) {
    up = 1 << (qstate->qubit_num - 1 - i);
    if (r >= qstate->prob_array[idx + up]) {
      idx = idx + up;
    }
  }
  
  for (i=0; i<mnum; i++) {
    mchar[i] = (idx >> (qstate->qubit_num - qid[i] - 1)) % 2;
  }

  SUC_RETURN(true);
}

static bool _qstate_measure_gpu(QState* qstate, int mnum, int* qid,
				char* measured_char, bool measure_update)
/* execute one shot measurement and update qstate according to measure_update flag */
{
  int			i, x;
  int			mval_qid     = 0;

  if (measure_update == true) { /* measure and update qstate */

    if (!(qstate_update_host_memory(qstate)))
      ERR_RETURN(ERROR_QSTATE_UPDATE_HOST_MEMORY, false);

    _qstate_get_measured_char_gpu(qstate, mnum, qid, measured_char);

    /* update qstate */
    for (i=0; i<mnum; i++) {
      mval_qid += ((int)measured_char[i] << (mnum - 1 - i));
    }
    for (i=0; i<qstate->state_num; i++) {
      if (!(select_bits(&x, i, mnum, qstate->qubit_num, qid)))
	ERR_RETURN(ERROR_INVALID_ARGUMENT, false);
      if (x != mval_qid) qstate->camp[i] = 0.0;
    }
    if (!(qstate_normalize(qstate))) ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

    if (!(qstate_update_device_memory(qstate)))
      ERR_RETURN(ERROR_QSTATE_UPDATE_DEVICE_MEMORY, false);

    qstate->prob_updated = false;
    qstate->d_prob_updated = false;
  }

  else { /* measure but not update qstate */
    _qstate_get_measured_char_gpu(qstate, mnum, qid, measured_char);
  }

  SUC_RETURN(true);
}

bool qstate_operate_qcirc_gpu(QState* qstate, CMem* cmem, QCirc* qcirc, bool measure_update)
/* one shot qcirc execution */
{
  QGate*		qgate	      = NULL;	/* quantum gate in quantum circuit */
  int                   i;
  int			dim	      = 0;
  COMPLEX*		U	      = NULL;
  hipDoubleComplex*	h_U	      = NULL;
  int                   q0	      = -1;
  int                   q1	      = -1;
  bool                  compo	      = false;	/* U is composite or not */
  int			mnum;
  int*			qid	      = NULL;
  int*			cid	      = NULL;
  bool			last;
  char*			measured_char = NULL;

  /* error check */
  if ((qstate == NULL || qcirc == NULL) ||
      (qstate->qubit_num < qcirc->qubit_num) ||
      (cmem != NULL && cmem->cmem_num < qcirc->cmem_num))
    ERR_RETURN(ERROR_INVALID_ARGUMENT,false);

  checkCudaErrors(hipHostMalloc((void**)&h_U, sizeof(hipDoubleComplex) * 16));

  /* malloc */
  if (cmem != NULL) {
    if (!(cid = (int*)malloc(sizeof(int) * cmem->cmem_num)))
      ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);

    if (!(measured_char = (char*)malloc(sizeof(int) * cmem->cmem_num)))
      ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);
  }
  if (!(qid = (int*)malloc(sizeof(int) * qstate->qubit_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);

  /* execute quantum circuit */
  qgate = qcirc->first;
  while (qgate != NULL) {

    if ((qgate->ctrl == -1) ||
	((qgate->ctrl != -1) && (cmem->bit_array[qgate->ctrl] == 1))) {

      /* unitary gate */
      if (kind_is_unitary(qgate->kind) == true) {

	if (!(qgate_get_next_unitary((void**)&qgate, qstate->gbank, &dim, &q0, &q1, (void**)&U, &compo))) {
	  ERR_RETURN(ERROR_GBANK_GET_UNITARY,false);
	}
	
	for (i=0; i<dim*dim; i++) {
	  h_U[i] = make_hipDoubleComplex(creal(U[i]), cimag(U[i]));
	}
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_U), h_U, sizeof(hipDoubleComplex) * dim * dim));

	if (compo == false && kind_is_controlled(qgate->kind) == true) {
	  if (!(_qstate_operate_controlled_gate_gpu_static(qstate, q0, q1))) {
	    ERR_RETURN(ERROR_INVALID_ARGUMENT,false);
	  }
	}
	else {
	  if (!(_qstate_operate_unitary_gpu_static(qstate, dim, q0, q1))) {
	    ERR_RETURN(ERROR_INVALID_ARGUMENT,false);
	  }
	}
	free(U); U = NULL;
	
	qgate = qgate->next;
      }
      /* reset */
      else if (kind_is_reset(qgate->kind) == true) {
      	if (!(qstate_reset(qstate, 1, qgate->qid)))
      	  ERR_RETURN(ERROR_CANT_RESET, false);
 	qgate = qgate->next;
      }
      /* measurement */
      else if (kind_is_measurement(qgate->kind) == true) {

	if (!(qgate_get_measurement_attributes((void**)&qgate, qstate->gbank, &mnum, qid, cid, &last))) {
	  ERR_RETURN(ERROR_QGATE_GET_NEXT_UNITARY, false);
	}
	if (!(_qstate_measure_gpu(qstate, mnum, qid, measured_char, measure_update)))
	  ERR_RETURN(ERROR_QSTATE_MEASURE, false);
	for (i=0; i<mnum; i++) {
	  cmem->bit_array[cid[i]] = measured_char[i];
	}
	qgate = qgate->next;
      }

      else {
      	ERR_RETURN(ERROR_QSTATE_OPERATE_QCIRC, false);
      }
    }
    else {
      qgate = qgate->next;
    }
  }

  checkCudaErrors(hipHostFree(h_U));

  SUC_RETURN(true);
}

bool qstate_operate_measure_gpu(QState* qstate, CMem* cmem, QCirc* qcirc,
				int shots, char* mchar_shots, bool out_state)
/* qcirc execution and get measurement data (suppose that qcirc includes only measurements) */
{
  char*			measured_char = NULL;
  int*			qid	      = NULL;
  int*			cid	      = NULL;
  QGate*		qgate	      = NULL;
  int			mnum;
  bool			last;
  int                   shots_tmp;
  bool			measure_update;
  bool			ans;
  float*		d_rand	      = NULL;
  hiprandGenerator_t	cugen;
  unsigned int          seed	      = (unsigned int)time(NULL);
  char*			d_mchar_shots = NULL;
  int			j,k;
  int			blocksize     = BLOCKSIZE;
  dim3			block (blocksize, 1, 1);
  dim3			grid ((shots + block.x - 1) / block.x, 1, 1);

  if ((qstate == NULL) || (cmem == NULL) || (qcirc == NULL) ||
      (shots < 1) || (mchar_shots == NULL) )
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);
  qcirc_is_measurement_only(qcirc, &ans);
  if (ans == false) ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  /* memory allocation */
  if (!(measured_char = (char*)malloc(sizeof(char) * qstate->qubit_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);
  if (!(cid = (int*)malloc(sizeof(int) * cmem->cmem_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);
  if (!(qid = (int*)malloc(sizeof(int) * qstate->qubit_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);

  checkCudaErrors(hipMalloc((void**)&(d_mchar_shots), sizeof(double) * cmem->cmem_num * shots));
  checkCudaErrors(hipMalloc((void**)&(d_rand), sizeof(int) * shots));

  /* random generator */
  hiprandCreateGenerator(&cugen, HIPRAND_RNG_PSEUDO_MTGP32);
  hiprandSetPseudoRandomGeneratorSeed(cugen, seed);
  hiprandGenerateUniform(cugen, (float*)d_rand, shots);
  
  /* measurement attributes */
  qgate = qcirc->first;
  if (!(qgate_get_measurement_attributes((void**)&qgate, qstate->gbank, &mnum, qid, cid, &last))) {
    ERR_RETURN(ERROR_QGATE_GET_NEXT_UNITARY, false);
  }

  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_qid), qid, sizeof(int) * MAX_QUBIT_NUM));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cid), cid, sizeof(int) * MAX_QUBIT_NUM));

  /* update prob array */
  _qstate_update_prob_array_gpu(qstate);

  if (out_state == true) shots_tmp = shots - 1;
  else shots_tmp = shots;
  
  /* shots_tmp times measurement -> not update qstate */
  cuda_qstate_operate_measure<<< grid, block >>>(qstate->d_camp, qstate->d_prob_array, d_mchar_shots,
						 d_rand, qstate->qubit_num, qstate->state_num,
						 cmem->cmem_num, mnum, shots_tmp);
  
  checkCudaErrors(hipMemcpy(mchar_shots, d_mchar_shots, sizeof(char) * cmem->cmem_num * shots,
			     hipMemcpyDeviceToHost));

  /* last one shot measurement -> update qstate */
  if (out_state == true) {
    measure_update = true;
    if (!(_qstate_measure_gpu(qstate, mnum, qid, measured_char, measure_update)))
      ERR_RETURN(ERROR_QSTATE_MEASURE, false);
    for (k=0; k<mnum; k++) {
      cmem->bit_array[cid[k]] = measured_char[k];
    }
    for (j=0; j<cmem->cmem_num; j++) {
      mchar_shots[(shots - 1) * cmem->cmem_num + j] = cmem->bit_array[j];
    }
  }
  
  if (out_state == true) {
    if (!(qstate_update_device_memory(qstate)))
      ERR_RETURN(ERROR_QSTATE_UPDATE_DEVICE_MEMORY, false);
  }

  qstate->d_prob_updated = false;

  hiprandDestroyGenerator(cugen);
  
  checkCudaErrors(hipFree(d_mchar_shots)); d_mchar_shots = NULL;
  checkCudaErrors(hipFree(d_rand)); d_rand = NULL;

  free(cid); cid = NULL;
  free(measured_char); measured_char = NULL;
  free(qid); qid = NULL;

  SUC_RETURN(true);
}

bool qstate_init_gpu(int qubit_num, void** qstate_out)
{
  QState		*qstate	   = NULL;
  int			 state_num = (1 << qubit_num);
  int			 blocksize = BLOCKSIZE;
  dim3			 block (blocksize, 1, 1);
  dim3			 grid ((state_num + block.x - 1) / block.x, 1, 1);
  hipDoubleComplex	 h_buf;

  if ((qubit_num < 1) || (qubit_num > MAX_QUBIT_NUM))
    ERR_RETURN(ERROR_OUT_OF_BOUND,false);
  
  if (!(qstate = (QState*)malloc(sizeof(QState))))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);

  qstate->qubit_num = qubit_num;
  qstate->state_num = state_num;
  qstate->use_gpu = true;

  /* allocate host memory */
  qstate->buf_id = 0;
  if (!(qstate->buffer_0 = (COMPLEX*)malloc(sizeof(COMPLEX) * state_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);
  if (!(qstate->buffer_1 = (COMPLEX*)malloc(sizeof(COMPLEX) * state_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);
  qstate->camp = qstate->buffer_0;

  if (!(qstate->prob_array = (double*)malloc(sizeof(double) * state_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);
  qstate->prob_updated = false;

  /* allocatie device memory */
  qstate->d_buf_id = 0;
  checkCudaErrors(hipMalloc((void**)&(qstate->d_buffer_0), sizeof(hipDoubleComplex) * state_num));
  
  checkCudaErrors(hipMalloc((void**)&(qstate->d_buffer_1), sizeof(hipDoubleComplex) * state_num));
  qstate->d_camp = qstate->d_buffer_0;

  checkCudaErrors(hipMalloc((void**)&(qstate->d_prob_array), sizeof(double) * state_num));
  qstate->d_prob_updated = false;

  /* initialize device memory */
  checkCudaErrors(hipMemset(qstate->d_buffer_0, 0, sizeof(hipDoubleComplex) * state_num));
  h_buf = make_hipDoubleComplex(1.0, 0.0);
  checkCudaErrors(hipMemcpy(qstate->d_buffer_0, &h_buf, sizeof(hipDoubleComplex),
			     hipMemcpyHostToDevice));

  /* set gbank */
  if (!(gbank_init((void**)&(qstate->gbank))))
      ERR_RETURN(ERROR_GBANK_INIT,false);

  *qstate_out = qstate;
  
  SUC_RETURN(true);
}

bool qstate_update_host_memory(QState* qstate)
{
  hipDoubleComplex*	h_camp = NULL;
  int			i;
  
  if (qstate == NULL) ERR_RETURN(ERROR_INVALID_ARGUMENT, false);
  if (qstate->use_gpu == false) SUC_RETURN(true);

  checkCudaErrors(hipHostMalloc((void**)&h_camp, sizeof(hipDoubleComplex) * qstate->state_num));

  checkCudaErrors(hipMemcpy(h_camp, qstate->d_camp, sizeof(hipDoubleComplex) * qstate->state_num,
			     hipMemcpyDeviceToHost));

  checkCudaErrors(hipMemcpy(qstate->prob_array, qstate->d_prob_array, sizeof(double) * qstate->state_num,
			     hipMemcpyDeviceToHost));
  qstate->prob_updated = qstate->d_prob_updated;

  for (i=0; i<qstate->state_num; i++) {
    qstate->camp[i] = h_camp[i].x + h_camp[i].y * COMP_I;
  }

  checkCudaErrors(hipHostFree(h_camp));

  SUC_RETURN(true);
}
  
bool qstate_update_device_memory(QState* qstate)
{
  hipDoubleComplex*	h_camp = NULL;
  int			i;
  
  if (qstate == NULL) ERR_RETURN(ERROR_INVALID_ARGUMENT, false);
  if (qstate->use_gpu == false) SUC_RETURN(true);

  checkCudaErrors(hipHostMalloc((void**)&h_camp, sizeof(hipDoubleComplex) * qstate->state_num));

  for (i=0; i<qstate->state_num; i++) {
    h_camp[i] = make_hipDoubleComplex(creal(qstate->camp[i]), cimag(qstate->camp[i]));
  }

  checkCudaErrors(hipMemcpy(qstate->d_camp, h_camp, sizeof(hipDoubleComplex) * qstate->state_num,
			     hipMemcpyHostToDevice));

  checkCudaErrors(hipMemcpy(qstate->d_prob_array, qstate->prob_array, sizeof(double) * qstate->state_num,
			     hipMemcpyHostToDevice));
  qstate->d_prob_updated = qstate->prob_updated;

  checkCudaErrors(hipHostFree(h_camp));

  SUC_RETURN(true);
}

void qstate_free_gpu(QState* qstate)
{
  if (qstate == NULL) return;
  
  if (qstate->buffer_0 != NULL) {
    free(qstate->buffer_0); qstate->buffer_0 = NULL;
  }
  if (qstate->buffer_1 != NULL) {
    free(qstate->buffer_1); qstate->buffer_1 = NULL;
  }
  if (qstate->gbank != NULL) {
    free(qstate->gbank); qstate->gbank = NULL;
  }

  checkCudaErrors(hipFree(qstate->d_buffer_0)); qstate->d_buffer_0 = NULL;
  checkCudaErrors(hipFree(qstate->d_buffer_1)); qstate->d_buffer_1 = NULL;
  
  free(qstate);
}
  
#ifdef __cplusplus
}
#endif
