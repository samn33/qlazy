#include "hip/hip_runtime.h"
/*
 *  qstate_gpu.cu
 */
#pragma GCC diagnostic ignored "-Wunused-but-set-variable"

#ifdef __cplusplus
extern "C" {
#endif

#include "qlazy.h"

#define BLOCKSIZE 32

__constant__ hipDoubleComplex d_U[16];

__global__ void cuda_qstate_operate_unitary2(hipDoubleComplex* d_camp_out, hipDoubleComplex* d_camp_in,
					     int qubit_num, int state_num, int n)
{
  int		nn, i, p, pp, sign, off;
  hipDoubleComplex	camp;

  nn = qubit_num - n - 1;
  
  i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < state_num) {
    
    p = (i >> nn) % 2;
    pp = p ^ 1;
    sign = (pp << 1) - 1; // pp=0 -> -1, pp=1 -> +1
    off = sign * (1 << nn);

    camp = hipCmul(d_U[IDX2(p,p)], d_camp_in[i]);
    d_camp_out[i] = hipCadd(camp, hipCmul(d_U[IDX2(p,pp)], d_camp_in[i + off]));
  }
}

__global__ void cuda_qstate_operate_unitary4(hipDoubleComplex* d_camp_out, hipDoubleComplex* d_camp_in,
					     int qubit_num, int state_num, int m, int n)
{
  int			mm, nn, i, p, pp, q, qq;
  int			l, sign_p, sign_q, off_p, off_q;
  hipDoubleComplex	camp;
  
  mm = qubit_num - m - 1;
  nn = qubit_num - n - 1;

  i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < state_num) {

    p = (i >> mm) % 2;
    pp = p ^ 1;
    q = (i >> nn) % 2;
    qq = q ^ 1;

    l = (p << 1) + q;

    sign_p = (pp << 1) - 1;
    sign_q = (qq << 1) - 1;

    off_p = sign_p * (1 << mm);
    off_q = sign_q * (1 << nn);

    camp = hipCmul(d_U[IDX4(l, l)], d_camp_in[i]);
    camp = hipCadd(camp, hipCmul(d_U[IDX4(l, (l^1))], d_camp_in[i + off_q]));
    camp = hipCadd(camp, hipCmul(d_U[IDX4(l, (l^2))], d_camp_in[i + off_p]));
    d_camp_out[i] = hipCadd(camp, hipCmul(d_U[IDX4(l, (l^3))], d_camp_in[i + off_q + off_p]));
  }
}

__global__ void cuda_qstate_operate_controlled_gate(hipDoubleComplex* d_camp_out, hipDoubleComplex* d_camp_in,
						    int qubit_num, int state_num, int m, int n)
{
  int			mm, nn, i, p, q, qq;
  int			l, sign_q, off_q;
  hipDoubleComplex	camp;
  
  mm = qubit_num - m - 1;
  nn = qubit_num - n - 1;

  i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < state_num) {

    p = (i >> mm) % 2;
    q = (i >> nn) % 2;
    qq = q ^ 1;

    l = (p << 1) + q;

    sign_q = (qq << 1) - 1;
    off_q = sign_q * (1 << nn);

    camp = hipCmul(d_U[IDX4(l, l)], d_camp_in[i]);
    d_camp_out[i] = hipCadd(camp, hipCmul(d_U[IDX4(l, (l^1))], d_camp_in[i + off_q]));
  }
}

static bool _qstate_operate_unitary_gpu_static(QState* qstate, int dim, int m, int n)
{
  int			qubit_num  = qstate->qubit_num;
  int			state_num  = qstate->state_num;
  hipDoubleComplex*	d_buffer_0 = qstate->d_buffer_0;
  hipDoubleComplex*	d_buffer_1 = qstate->d_buffer_1;
  int			blocksize  = BLOCKSIZE;
  dim3			block (blocksize, 1, 1);
  dim3			grid ((state_num + block.x - 1) / block.x, 1, 1);

  if ((qstate == NULL) || (qstate->use_gpu == false) || (dim < 0))
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  if (dim == 2) {
    /* 0 -> 1 */
    if (qstate->d_buf_id == 0) {
      cuda_qstate_operate_unitary2<<< grid, block >>>(d_buffer_1, d_buffer_0,
      						      qubit_num, state_num, m);
      qstate->d_buf_id = 1;
      qstate->d_camp = qstate->d_buffer_1;
    }
    /* 1 -> 0 */
    else {
      cuda_qstate_operate_unitary2<<< grid, block >>>(d_buffer_0, d_buffer_1,
						      qubit_num, state_num, m);
      qstate->d_buf_id = 0;
      qstate->d_camp = qstate->d_buffer_0;
    }
  }
  else if (dim == 4) {
    /* 0 -> 1 */
    if (qstate->d_buf_id == 0) {
      cuda_qstate_operate_unitary4<<< grid, block >>>(d_buffer_1, d_buffer_0,
      						      qubit_num, state_num, m, n);
      qstate->d_buf_id = 1;
      qstate->d_camp = qstate->d_buffer_1;
    }
    /* 1 -> 0 */
    else {
      cuda_qstate_operate_unitary4<<< grid, block >>>(d_buffer_0, d_buffer_1,
						      qubit_num, state_num, m, n);
      qstate->d_buf_id = 0;
      qstate->d_camp = qstate->d_buffer_0;
    }
  }
  else {
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);
  }

  SUC_RETURN(true);
}

static bool _qstate_operate_controlled_gate_gpu_static(QState* qstate, int m, int n)
{
  int			qubit_num  = qstate->qubit_num;
  int			state_num  = qstate->state_num;
  hipDoubleComplex*	d_buffer_0 = qstate->d_buffer_0;
  hipDoubleComplex*	d_buffer_1 = qstate->d_buffer_1;
  int			blocksize  = BLOCKSIZE;
  dim3			block (blocksize, 1, 1);
  dim3			grid ((state_num + block.x - 1) / block.x, 1, 1);

  if ((qstate == NULL) || (qstate->use_gpu == false))
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  /* 0 -> 1 */
  if (qstate->d_buf_id == 0) {
    cuda_qstate_operate_controlled_gate<<< grid, block >>>(d_buffer_1, d_buffer_0,
							   qubit_num, state_num, m, n);
    qstate->d_buf_id = 1;
    qstate->d_camp = qstate->d_buffer_1;
  }
  /* 1 -> 0 */
  else {
    cuda_qstate_operate_controlled_gate<<< grid, block >>>(d_buffer_0, d_buffer_1,
							   qubit_num, state_num, m, n);
    qstate->d_buf_id = 0;
    qstate->d_camp = qstate->d_buffer_0;
  }

  SUC_RETURN(true);
}

bool qstate_operate_controlled_gate_gpu(QState* qstate, COMPLEX* U, int m, int n)
{
  int			i;
  hipDoubleComplex*	h_U = NULL;

  if ((qstate == NULL) || (qstate->use_gpu == false) || (U == NULL))
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  checkCudaErrors(hipHostMalloc((void**)&h_U, sizeof(hipDoubleComplex) * 16));

  for (i=0; i<16; i++) {
    h_U[i] = make_hipDoubleComplex(creal(U[i]), cimag(U[i]));
  }
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_U), h_U, sizeof(hipDoubleComplex) * 16));

  _qstate_operate_controlled_gate_gpu_static(qstate, m, n);

  checkCudaErrors(hipHostFree(h_U));

  SUC_RETURN(true);
}

bool qstate_operate_unitary_gpu(QState* qstate, COMPLEX* U, int dim, int m, int n)
{
  int			i;
  hipDoubleComplex*	h_U = NULL;

  if ((qstate == NULL) || (qstate->use_gpu == false) || (U == NULL) || (dim < 0))
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  checkCudaErrors(hipHostMalloc((void**)&h_U, sizeof(hipDoubleComplex) * 16));

  for (i=0; i<dim*dim; i++) {
    h_U[i] = make_hipDoubleComplex(creal(U[i]), cimag(U[i]));
  }
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_U), h_U, sizeof(hipDoubleComplex) * dim * dim));

  _qstate_operate_unitary_gpu_static(qstate, dim, m, n);

  checkCudaErrors(hipHostFree(h_U));

  SUC_RETURN(true);
}

bool qstate_operate_qcirc_gpu(QState* qstate, CMem* cmem, QCirc* qcirc)
{
  QGate*		qgate = NULL;	/* quantum gate in quantum circuit */
  double		angle = 0.0;	/* measurement angle */
  double		phase = 0.0;	/* measurement phase */
  int			qubit_id[MAX_QUBIT_NUM];
  int			mes_id;
  int                   i;
  int			dim   = 0;
  COMPLEX*		U     = NULL;
  hipDoubleComplex*	h_U   = NULL;
  int                   q0 = -1;
  int                   q1 = -1;
  bool                  compo = false;  /* U is composite or not */

  /* error check */
  if ((qstate == NULL || qcirc == NULL) ||
      (qstate->qubit_num < qcirc->qubit_num) ||
      (cmem != NULL && cmem->cmem_num < qcirc->cmem_num))
    ERR_RETURN(ERROR_INVALID_ARGUMENT,false);

  checkCudaErrors(hipHostMalloc((void**)&h_U, sizeof(hipDoubleComplex) * 16));

  qgate = qcirc->first;
  while (qgate != NULL) {

    if ((qgate->ctrl == -1) ||
	((qgate->ctrl != -1) && (cmem->bit_array[qgate->ctrl] == 1))) {

      /* unitary gate */
      if (kind_is_unitary(qgate->kind) == true) {

	if (!(qgate_get_next_unitary((void**)&qgate, qstate->gbank, &dim, &q0, &q1, (void**)&U, &compo))) {
	  ERR_RETURN(ERROR_GBANK_GET_UNITARY,false);
	}
	
	for (i=0; i<dim*dim; i++) {
	  h_U[i] = make_hipDoubleComplex(creal(U[i]), cimag(U[i]));
	}
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_U), h_U, sizeof(hipDoubleComplex) * dim * dim));

	if (compo == false && kind_is_controlled(qgate->kind) == true) {
	  if (!(_qstate_operate_controlled_gate_gpu_static(qstate, q0, q1))) {
	    ERR_RETURN(ERROR_INVALID_ARGUMENT,false);
	  }
	}
	else {
	  if (!(_qstate_operate_unitary_gpu_static(qstate, dim, q0, q1))) {
	    ERR_RETURN(ERROR_INVALID_ARGUMENT,false);
	  }
	}
	free(U); U = NULL;
	
	qgate = qgate->next;
      }
      /* reset */
      else if (kind_is_reset(qgate->kind) == true) {
      	if (!(qstate_reset(qstate, 1, qgate->qid)))
      	  ERR_RETURN(ERROR_CANT_RESET, false);
 	qgate = qgate->next;
      }
      /* measurement */
      else if (kind_is_measurement(qgate->kind) == true) {
      	qubit_id[0] = qgate->qid[0];
      	if (!(qstate_measure(qstate, angle, phase, 1, qubit_id, &mes_id)))
      	  ERR_RETURN(ERROR_QSTATE_MEASURE, false);
      	if (mes_id < 0 || mes_id > 1) ERR_RETURN(ERROR_QSTATE_MEASURE, false);
      	if (qgate->c != -1) cmem->bit_array[qgate->c] = (BYTE)mes_id; /* measured value is stored to classical register */
	qgate = qgate->next;
      }
      else {
      	ERR_RETURN(ERROR_QSTATE_OPERATE_QCIRC, false);
      }
    }
    else {
      qgate = qgate->next;
    }
  }

  checkCudaErrors(hipHostFree(h_U));

  SUC_RETURN(true);
}

bool qstate_init_gpu(int qubit_num, void** qstate_out)
{
  QState		*qstate	   = NULL;
  int			 state_num = (1 << qubit_num);
  int			 blocksize = BLOCKSIZE;
  dim3			 block (blocksize, 1, 1);
  dim3			 grid ((state_num + block.x - 1) / block.x, 1, 1);
  hipDoubleComplex	 h_buf;

  if ((qubit_num < 1) || (qubit_num > MAX_QUBIT_NUM))
    ERR_RETURN(ERROR_OUT_OF_BOUND,false);
  
  if (!(qstate = (QState*)malloc(sizeof(QState))))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);

  qstate->qubit_num = qubit_num;
  qstate->state_num = state_num;
  qstate->use_gpu = true;

  /* host memory */
  qstate->buf_id = 0;
  if (!(qstate->buffer_0 = (COMPLEX*)malloc(sizeof(COMPLEX) * state_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY,false);
  if (!(qstate->buffer_1 = (COMPLEX*)malloc(sizeof(COMPLEX) * state_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY,false);
  qstate->camp = qstate->buffer_0;

  /* allocatie device memory */
  qstate->d_buf_id = 0;
  checkCudaErrors(hipMalloc((void**)&(qstate->d_buffer_0), sizeof(hipDoubleComplex) * state_num));
  
  checkCudaErrors(hipMalloc((void**)&(qstate->d_buffer_1), sizeof(hipDoubleComplex) * state_num));
  qstate->d_camp = qstate->d_buffer_0;

  /* initialize device memory */
  checkCudaErrors(hipMemset(qstate->d_buffer_0, 0, sizeof(hipDoubleComplex) * state_num));
  h_buf = make_hipDoubleComplex(1.0, 0.0);
  checkCudaErrors(hipMemcpy(qstate->d_buffer_0, &h_buf, sizeof(hipDoubleComplex),
			     hipMemcpyHostToDevice));

  /* set gbank */
  if (!(gbank_init((void**)&(qstate->gbank))))
      ERR_RETURN(ERROR_GBANK_INIT,false);

  *qstate_out = qstate;
  
  SUC_RETURN(true);
}

bool qstate_update_host_memory(QState* qstate)
{
  hipDoubleComplex*	h_camp = NULL;
  int			i;
  
  if (qstate == NULL) ERR_RETURN(ERROR_INVALID_ARGUMENT, false);
  if (qstate->use_gpu == false) SUC_RETURN(true);

  checkCudaErrors(hipHostMalloc((void**)&h_camp, sizeof(hipDoubleComplex) * qstate->state_num));

  checkCudaErrors(hipMemcpy(h_camp, qstate->d_camp, sizeof(hipDoubleComplex) * qstate->state_num,
			     hipMemcpyDeviceToHost));

  for (i=0; i<qstate->state_num; i++) {
    qstate->camp[i] = h_camp[i].x + h_camp[i].y * COMP_I;
  }

  checkCudaErrors(hipHostFree(h_camp));

  SUC_RETURN(true);
}
  
bool qstate_update_device_memory(QState* qstate)
{
  hipDoubleComplex*	h_camp = NULL;
  int			i;
  
  if (qstate == NULL) ERR_RETURN(ERROR_INVALID_ARGUMENT, false);
  if (qstate->use_gpu == false) SUC_RETURN(true);

  checkCudaErrors(hipHostMalloc((void**)&h_camp, sizeof(hipDoubleComplex) * qstate->state_num));

  for (i=0; i<qstate->state_num; i++) {
    h_camp[i] = make_hipDoubleComplex(creal(qstate->camp[i]), cimag(qstate->camp[i]));
  }

  checkCudaErrors(hipMemcpy(qstate->d_camp, h_camp, sizeof(hipDoubleComplex) * qstate->state_num,
			     hipMemcpyHostToDevice));

  checkCudaErrors(hipHostFree(h_camp));

  SUC_RETURN(true);
}

void qstate_free_gpu(QState* qstate)
{
  if (qstate == NULL) return;
  
  if (qstate->buffer_0 != NULL) {
    free(qstate->buffer_0); qstate->buffer_0 = NULL;
  }
  if (qstate->buffer_1 != NULL) {
    free(qstate->buffer_1); qstate->buffer_1 = NULL;
  }
  if (qstate->gbank != NULL) {
    free(qstate->gbank); qstate->gbank = NULL;
  }

  checkCudaErrors(hipFree(qstate->d_buffer_0)); qstate->d_buffer_0 = NULL;
  checkCudaErrors(hipFree(qstate->d_buffer_1)); qstate->d_buffer_1 = NULL;
  
  free(qstate);
}
  
#ifdef __cplusplus
}
#endif
