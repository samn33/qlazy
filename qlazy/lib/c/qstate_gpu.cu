#include "hip/hip_runtime.h"
/*
 *  qstate_gpu.cu
 */
#pragma GCC diagnostic ignored "-Wunused-but-set-variable"

#ifdef __cplusplus
extern "C" {
#endif

#include "qlazy.h"

#define BLOCKSIZE 32

__constant__ hipDoubleComplex d_U[16];

__global__ void cuda_qstate_operate_unitary2(hipDoubleComplex* d_camp_out, hipDoubleComplex* d_camp_in,
					     int qubit_num, int state_num, int n)
{
  int		nn, i, p, pp, sign, off;
  hipDoubleComplex	camp;

  nn = qubit_num - n - 1;
  
  i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < state_num) {
    
    p = (i >> nn) % 2;
    pp = p ^ 1;
    sign = (pp << 1) - 1; // pp=0 -> -1, pp=1 -> +1
    off = sign * (1 << nn);

    camp = hipCmul(d_U[IDX2(p,p)], d_camp_in[i]);
    d_camp_out[i] = hipCadd(camp, hipCmul(d_U[IDX2(p,pp)], d_camp_in[i + off]));
  }
}

__global__ void cuda_qstate_operate_unitary4(hipDoubleComplex* d_camp_out, hipDoubleComplex* d_camp_in,
					     int qubit_num, int state_num, int m, int n)
{
  int			mm, nn, i, p, pp, q, qq;
  int			l, sign_p, sign_q, off_p, off_q;
  hipDoubleComplex	camp;
  
  mm = qubit_num - m - 1;
  nn = qubit_num - n - 1;

  i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < state_num) {

    p = (i >> mm) % 2;
    pp = p ^ 1;
    q = (i >> nn) % 2;
    qq = q ^ 1;

    l = (p << 1) + q;

    sign_p = (pp << 1) - 1;
    sign_q = (qq << 1) - 1;

    off_p = sign_p * (1 << mm);
    off_q = sign_q * (1 << nn);

    camp = hipCmul(d_U[IDX4(l, l)], d_camp_in[i]);
    camp = hipCadd(camp, hipCmul(d_U[IDX4(l, (l^1))], d_camp_in[i + off_q]));
    camp = hipCadd(camp, hipCmul(d_U[IDX4(l, (l^2))], d_camp_in[i + off_p]));
    d_camp_out[i] = hipCadd(camp, hipCmul(d_U[IDX4(l, (l^3))], d_camp_in[i + off_q + off_p]));
  }
}

__global__ void cuda_qstate_operate_controlled_gate(hipDoubleComplex* d_camp_out, hipDoubleComplex* d_camp_in,
						    int qubit_num, int state_num, int m, int n)
{
  int			mm, nn, i, p, q, qq;
  int			l, sign_q, off_q;
  hipDoubleComplex	camp;
  
  mm = qubit_num - m - 1;
  nn = qubit_num - n - 1;

  i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < state_num) {

    p = (i >> mm) % 2;
    q = (i >> nn) % 2;
    qq = q ^ 1;

    l = (p << 1) + q;

    sign_q = (qq << 1) - 1;
    off_q = sign_q * (1 << nn);

    camp = hipCmul(d_U[IDX4(l, l)], d_camp_in[i]);
    d_camp_out[i] = hipCadd(camp, hipCmul(d_U[IDX4(l, (l^1))], d_camp_in[i + off_q]));
  }
}

static bool _qstate_operate_unitary_gpu_static(QState* qstate, int dim, int m, int n)
{
  int			qubit_num  = qstate->qubit_num;
  int			state_num  = qstate->state_num;
  hipDoubleComplex*	d_buffer_0 = qstate->d_buffer_0;
  hipDoubleComplex*	d_buffer_1 = qstate->d_buffer_1;
  int			blocksize  = BLOCKSIZE;
  dim3			block (blocksize, 1, 1);
  dim3			grid ((state_num + block.x - 1) / block.x, 1, 1);

  if ((qstate == NULL) || (qstate->use_gpu == false) || (dim < 0))
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  if (dim == 2) {
    /* 0 -> 1 */
    if (qstate->d_buf_id == 0) {
      cuda_qstate_operate_unitary2<<< grid, block >>>(d_buffer_1, d_buffer_0,
      						      qubit_num, state_num, m);
      qstate->d_buf_id = 1;
      qstate->d_camp = qstate->d_buffer_1;
    }
    /* 1 -> 0 */
    else {
      cuda_qstate_operate_unitary2<<< grid, block >>>(d_buffer_0, d_buffer_1,
						      qubit_num, state_num, m);
      qstate->d_buf_id = 0;
      qstate->d_camp = qstate->d_buffer_0;
    }
  }
  else if (dim == 4) {
    /* 0 -> 1 */
    if (qstate->d_buf_id == 0) {
      cuda_qstate_operate_unitary4<<< grid, block >>>(d_buffer_1, d_buffer_0,
      						      qubit_num, state_num, m, n);
      qstate->d_buf_id = 1;
      qstate->d_camp = qstate->d_buffer_1;
    }
    /* 1 -> 0 */
    else {
      cuda_qstate_operate_unitary4<<< grid, block >>>(d_buffer_0, d_buffer_1,
						      qubit_num, state_num, m, n);
      qstate->d_buf_id = 0;
      qstate->d_camp = qstate->d_buffer_0;
    }
  }
  else {
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);
  }

  qstate->d_prob_updated = false;

  SUC_RETURN(true);
}

static bool _qstate_operate_controlled_gate_gpu_static(QState* qstate, int m, int n)
{
  int			qubit_num  = qstate->qubit_num;
  int			state_num  = qstate->state_num;
  hipDoubleComplex*	d_buffer_0 = qstate->d_buffer_0;
  hipDoubleComplex*	d_buffer_1 = qstate->d_buffer_1;
  int			blocksize  = BLOCKSIZE;
  dim3			block (blocksize, 1, 1);
  dim3			grid ((state_num + block.x - 1) / block.x, 1, 1);

  if ((qstate == NULL) || (qstate->use_gpu == false))
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  /* 0 -> 1 */
  if (qstate->d_buf_id == 0) {
    cuda_qstate_operate_controlled_gate<<< grid, block >>>(d_buffer_1, d_buffer_0,
							   qubit_num, state_num, m, n);
    qstate->d_buf_id = 1;
    qstate->d_camp = qstate->d_buffer_1;
  }
  /* 1 -> 0 */
  else {
    cuda_qstate_operate_controlled_gate<<< grid, block >>>(d_buffer_0, d_buffer_1,
							   qubit_num, state_num, m, n);
    qstate->d_buf_id = 0;
    qstate->d_camp = qstate->d_buffer_0;
  }

  qstate->d_prob_updated = false;

  SUC_RETURN(true);
}

bool qstate_operate_controlled_gate_gpu(QState* qstate, COMPLEX* U, int m, int n)
{
  int			i;
  hipDoubleComplex*	h_U = NULL;

  if ((qstate == NULL) || (qstate->use_gpu == false) || (U == NULL))
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  checkCudaErrors(hipHostMalloc((void**)&h_U, sizeof(hipDoubleComplex) * 16));

  for (i=0; i<16; i++) {
    h_U[i] = make_hipDoubleComplex(creal(U[i]), cimag(U[i]));
  }
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_U), h_U, sizeof(hipDoubleComplex) * 16));

  _qstate_operate_controlled_gate_gpu_static(qstate, m, n);

  checkCudaErrors(hipHostFree(h_U));

  SUC_RETURN(true);
}

bool qstate_operate_unitary_gpu(QState* qstate, COMPLEX* U, int dim, int m, int n)
{
  int			i;
  hipDoubleComplex*	h_U = NULL;

  if ((qstate == NULL) || (qstate->use_gpu == false) || (U == NULL) || (dim < 0))
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  checkCudaErrors(hipHostMalloc((void**)&h_U, sizeof(hipDoubleComplex) * 16));

  for (i=0; i<dim*dim; i++) {
    h_U[i] = make_hipDoubleComplex(creal(U[i]), cimag(U[i]));
  }
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_U), h_U, sizeof(hipDoubleComplex) * dim * dim));

  _qstate_operate_unitary_gpu_static(qstate, dim, m, n);

  checkCudaErrors(hipHostFree(h_U));

  SUC_RETURN(true);
}

__global__ void cuda_qstate_update_prob_array(hipDoubleComplex* d_camp, double* d_prob_array,
					      int state_num)
{
  double p = 0.0;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < state_num) {
    p = hipCabs(d_camp[i]);
    d_prob_array[i] = p * p;
  }
}

static bool _qstate_get_measured_char_gpu(QState* qstate, int mnum, int* qid, char* mchar)
/* not update qstate, get measured char only */
{
  hipDoubleComplex*	d_buffer_0   = qstate->d_buffer_0;
  hipDoubleComplex*	d_buffer_1   = qstate->d_buffer_1;
  hipDoubleComplex*      d_camp	     = NULL;
  double*		d_prob_array = qstate->d_prob_array;
  double		r	     = 0.0;
  double		prob_s	     = 0.0;
  double		prob_e	     = 0.0;
  int			value	     = 0;
  int			bit	     = 0;
  int			blocksize    = BLOCKSIZE;
  dim3			block (blocksize, 1, 1);
  dim3			grid ((qstate->state_num + block.x - 1) / block.x, 1, 1);
  int			i;

  if (qstate == NULL) ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

  /* update prob_array, qstate */
  if (qstate->d_prob_updated == false) {
    if (qstate->d_buf_id == 0) d_camp = d_buffer_0;
    else d_camp = d_buffer_1;

    cuda_qstate_update_prob_array<<< grid, block >>>(d_camp, d_prob_array,
						     qstate->state_num);
    qstate->d_prob_updated = true;

    if (!(qstate_update_host_memory(qstate)))
      ERR_RETURN(ERROR_QSTATE_UPDATE_HOST_MEMORY, false);
  }
  
  r = rand() / (double)RAND_MAX;
  for (i=0; i<qstate->state_num; i++) {
    prob_s = prob_e;
    //prob_e = qstate->prob_array[i];
    prob_e = prob_s + qstate->prob_array[i];
    if (r >= prob_s && r < prob_e) {
      value = i;
      break;
    }
  }

  for (i=0; i<mnum; i++) {
    bit = (value >> (qstate->qubit_num - qid[i] - 1)) % 2;
    if (bit == 0) mchar[i] = 0;
    else mchar[i] = 1;
  }

  SUC_RETURN(true);
}

static bool _qstate_measure_gpu(QState* qstate, int mnum, int* qid,
				char* measured_char, bool measure_update)
/* ececute one shot measurement and update qstate according to measure_update flag */
{
  int			i, x;
  int			mval_qid     = 0;

  if (measure_update == true) { /* measure and update qstate */
    _qstate_get_measured_char_gpu(qstate, mnum, qid, measured_char);

    /* update qstate */
    for (i=0; i<mnum; i++) {
      mval_qid += ((int)measured_char[i] << (mnum - 1 - i));
    }
    for (i=0; i<qstate->state_num; i++) {
      if (!(select_bits(&x, i, mnum, qstate->qubit_num, qid)))
	ERR_RETURN(ERROR_INVALID_ARGUMENT, false);
      if (x != mval_qid) qstate->camp[i] = 0.0;
    }
    if (!(qstate_normalize(qstate))) ERR_RETURN(ERROR_INVALID_ARGUMENT, false);

    if (!(qstate_update_device_memory(qstate)))
      ERR_RETURN(ERROR_QSTATE_UPDATE_DEVICE_MEMORY, false);

    qstate->prob_updated = false;
    qstate->d_prob_updated = false;
  }

  else { /* measure but not update qstate */
    _qstate_get_measured_char_gpu(qstate, mnum, qid, measured_char);
  }

  SUC_RETURN(true);
}

bool qstate_operate_qcirc_gpu(QState* qstate, CMem* cmem, QCirc* qcirc, bool measure_update)
/* one shot qcirc execution */
{
  QGate*		qgate	      = NULL;	/* quantum gate in quantum circuit */
  int                   i;
  int			dim	      = 0;
  COMPLEX*		U	      = NULL;
  hipDoubleComplex*	h_U	      = NULL;
  int                   q0	      = -1;
  int                   q1	      = -1;
  bool                  compo	      = false;	/* U is composite or not */
  int			mnum;
  int*			qid	      = NULL;
  int*			cid	      = NULL;
  bool			last;
  char*			measured_char = NULL;

  /* error check */
  if ((qstate == NULL || qcirc == NULL) ||
      (qstate->qubit_num < qcirc->qubit_num) ||
      (cmem != NULL && cmem->cmem_num < qcirc->cmem_num))
    ERR_RETURN(ERROR_INVALID_ARGUMENT,false);

  checkCudaErrors(hipHostMalloc((void**)&h_U, sizeof(hipDoubleComplex) * 16));

  /* malloc */
  if (cmem != NULL) {
    if (!(cid = (int*)malloc(sizeof(int) * cmem->cmem_num)))
      ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);

    if (!(measured_char = (char*)malloc(sizeof(int) * cmem->cmem_num)))
      ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);
  }
  if (!(qid = (int*)malloc(sizeof(int) * qstate->qubit_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);

  /* execute quantum circuit */
  qgate = qcirc->first;
  while (qgate != NULL) {

    if ((qgate->ctrl == -1) ||
	((qgate->ctrl != -1) && (cmem->bit_array[qgate->ctrl] == 1))) {

      /* unitary gate */
      if (kind_is_unitary(qgate->kind) == true) {

	if (!(qgate_get_next_unitary((void**)&qgate, qstate->gbank, &dim, &q0, &q1, (void**)&U, &compo))) {
	  ERR_RETURN(ERROR_GBANK_GET_UNITARY,false);
	}
	
	for (i=0; i<dim*dim; i++) {
	  h_U[i] = make_hipDoubleComplex(creal(U[i]), cimag(U[i]));
	}
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_U), h_U, sizeof(hipDoubleComplex) * dim * dim));

	if (compo == false && kind_is_controlled(qgate->kind) == true) {
	  if (!(_qstate_operate_controlled_gate_gpu_static(qstate, q0, q1))) {
	    ERR_RETURN(ERROR_INVALID_ARGUMENT,false);
	  }
	}
	else {
	  if (!(_qstate_operate_unitary_gpu_static(qstate, dim, q0, q1))) {
	    ERR_RETURN(ERROR_INVALID_ARGUMENT,false);
	  }
	}
	free(U); U = NULL;
	
	qgate = qgate->next;
      }
      /* reset */
      else if (kind_is_reset(qgate->kind) == true) {
      	if (!(qstate_reset(qstate, 1, qgate->qid)))
      	  ERR_RETURN(ERROR_CANT_RESET, false);
 	qgate = qgate->next;
      }
      /* measurement */
      else if (kind_is_measurement(qgate->kind) == true) {

	if (!(qgate_get_measurement_attributes((void**)&qgate, qstate->gbank, &mnum, qid, cid, &last))) {
	  ERR_RETURN(ERROR_QGATE_GET_NEXT_UNITARY, false);
	}
	if (!(_qstate_measure_gpu(qstate, mnum, qid, measured_char, measure_update)))
	  ERR_RETURN(ERROR_QSTATE_MEASURE, false);
	for (i=0; i<mnum; i++) {
	  cmem->bit_array[cid[i]] = measured_char[i];
	}
	qgate = qgate->next;
      }

      else {
      	ERR_RETURN(ERROR_QSTATE_OPERATE_QCIRC, false);
      }
    }
    else {
      qgate = qgate->next;
    }
  }

  checkCudaErrors(hipHostFree(h_U));

  SUC_RETURN(true);
}

bool qstate_operate_measure_gpu(QState* qstate, CMem* cmem, QCirc* qcirc, int shots, char* mchar_shots)
/* qcirc execution and get measurement data (suppose that qcirc includes only measurements) */
{
  char*		measured_char = NULL;
  int*		qid	      = NULL;
  int*		cid	      = NULL;
  QGate*	qgate	      = NULL;
  int		mnum;
  bool		last;
  bool		measure_update;
  bool		ans;
  int		i,j,k;

  if ((qstate == NULL) || (cmem == NULL) || (qcirc == NULL) ||
      (shots < 1) || (mchar_shots == NULL) )
    ERR_RETURN(ERROR_INVALID_ARGUMENT, false);
  qcirc_is_measurement_only(qcirc, &ans);
  if (ans == false) ERR_RETURN(ERROR_INVALID_ARGUMENT, false);
  
  if (!(measured_char = (char*)malloc(sizeof(char) * qstate->qubit_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);
  if (!(cid = (int*)malloc(sizeof(int) * cmem->cmem_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);
  if (!(qid = (int*)malloc(sizeof(int) * qstate->qubit_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);

  qgate = qcirc->first;
  if (!(qgate_get_measurement_attributes((void**)&qgate, qstate->gbank, &mnum, qid, cid, &last))) {
    ERR_RETURN(ERROR_QGATE_GET_NEXT_UNITARY, false);
  }

  // TODO: -> kernel function
  measure_update = false;
  for (i=0; i<shots; i++) {
    if (i < shots - 1) measure_update = false;
    else measure_update = true;

    if (!(_qstate_measure_gpu(qstate, mnum, qid, measured_char, measure_update)))
      ERR_RETURN(ERROR_QSTATE_MEASURE, false);
    for (k=0; k<mnum; k++) {
      cmem->bit_array[cid[k]] = measured_char[k];
    }
    for (j=0; j<cmem->cmem_num; j++) {
      mchar_shots[i * cmem->cmem_num + j] = cmem->bit_array[j];
    }
  }

  free(cid); cid = NULL;
  free(measured_char); measured_char = NULL;
  free(qid); qid = NULL;

  SUC_RETURN(true);
}

bool qstate_init_gpu(int qubit_num, void** qstate_out)
{
  QState		*qstate	   = NULL;
  int			 state_num = (1 << qubit_num);
  int			 blocksize = BLOCKSIZE;
  dim3			 block (blocksize, 1, 1);
  dim3			 grid ((state_num + block.x - 1) / block.x, 1, 1);
  hipDoubleComplex	 h_buf;

  if ((qubit_num < 1) || (qubit_num > MAX_QUBIT_NUM))
    ERR_RETURN(ERROR_OUT_OF_BOUND,false);
  
  if (!(qstate = (QState*)malloc(sizeof(QState))))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);

  qstate->qubit_num = qubit_num;
  qstate->state_num = state_num;
  qstate->use_gpu = true;

  /* allocate host memory */
  qstate->buf_id = 0;
  if (!(qstate->buffer_0 = (COMPLEX*)malloc(sizeof(COMPLEX) * state_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);
  if (!(qstate->buffer_1 = (COMPLEX*)malloc(sizeof(COMPLEX) * state_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);
  qstate->camp = qstate->buffer_0;

  if (!(qstate->prob_array = (double*)malloc(sizeof(double) * state_num)))
    ERR_RETURN(ERROR_CANT_ALLOC_MEMORY, false);
  qstate->prob_updated = false;

  /* allocatie device memory */
  qstate->d_buf_id = 0;
  checkCudaErrors(hipMalloc((void**)&(qstate->d_buffer_0), sizeof(hipDoubleComplex) * state_num));
  
  checkCudaErrors(hipMalloc((void**)&(qstate->d_buffer_1), sizeof(hipDoubleComplex) * state_num));
  qstate->d_camp = qstate->d_buffer_0;

  checkCudaErrors(hipMalloc((void**)&(qstate->d_prob_array), sizeof(double) * state_num));
  qstate->d_prob_updated = false;

  /* initialize device memory */
  checkCudaErrors(hipMemset(qstate->d_buffer_0, 0, sizeof(hipDoubleComplex) * state_num));
  h_buf = make_hipDoubleComplex(1.0, 0.0);
  checkCudaErrors(hipMemcpy(qstate->d_buffer_0, &h_buf, sizeof(hipDoubleComplex),
			     hipMemcpyHostToDevice));

  /* set gbank */
  if (!(gbank_init((void**)&(qstate->gbank))))
      ERR_RETURN(ERROR_GBANK_INIT,false);

  *qstate_out = qstate;
  
  SUC_RETURN(true);
}

bool qstate_update_host_memory(QState* qstate)
{
  hipDoubleComplex*	h_camp = NULL;
  int			i;
  
  if (qstate == NULL) ERR_RETURN(ERROR_INVALID_ARGUMENT, false);
  if (qstate->use_gpu == false) SUC_RETURN(true);

  checkCudaErrors(hipHostMalloc((void**)&h_camp, sizeof(hipDoubleComplex) * qstate->state_num));

  checkCudaErrors(hipMemcpy(h_camp, qstate->d_camp, sizeof(hipDoubleComplex) * qstate->state_num,
			     hipMemcpyDeviceToHost));

  checkCudaErrors(hipMemcpy(qstate->prob_array, qstate->d_prob_array, sizeof(double) * qstate->state_num,
			     hipMemcpyDeviceToHost));
  // qstate->prob_updated = true;
  // qstate->d_prob_updated = true;
  qstate->prob_updated = qstate->d_prob_updated;

  for (i=0; i<qstate->state_num; i++) {
    qstate->camp[i] = h_camp[i].x + h_camp[i].y * COMP_I;
  }

  checkCudaErrors(hipHostFree(h_camp));

  SUC_RETURN(true);
}
  
bool qstate_update_device_memory(QState* qstate)
{
  hipDoubleComplex*	h_camp = NULL;
  int			i;
  
  if (qstate == NULL) ERR_RETURN(ERROR_INVALID_ARGUMENT, false);
  if (qstate->use_gpu == false) SUC_RETURN(true);

  checkCudaErrors(hipHostMalloc((void**)&h_camp, sizeof(hipDoubleComplex) * qstate->state_num));

  for (i=0; i<qstate->state_num; i++) {
    h_camp[i] = make_hipDoubleComplex(creal(qstate->camp[i]), cimag(qstate->camp[i]));
  }

  checkCudaErrors(hipMemcpy(qstate->d_camp, h_camp, sizeof(hipDoubleComplex) * qstate->state_num,
			     hipMemcpyHostToDevice));

  checkCudaErrors(hipMemcpy(qstate->d_prob_array, qstate->prob_array, sizeof(double) * qstate->state_num,
			     hipMemcpyHostToDevice));
  //qstate->prob_updated = true;
  //qstate->d_prob_updated = true;
  qstate->d_prob_updated = qstate->prob_updated;

  checkCudaErrors(hipHostFree(h_camp));

  SUC_RETURN(true);
}

void qstate_free_gpu(QState* qstate)
{
  if (qstate == NULL) return;
  
  if (qstate->buffer_0 != NULL) {
    free(qstate->buffer_0); qstate->buffer_0 = NULL;
  }
  if (qstate->buffer_1 != NULL) {
    free(qstate->buffer_1); qstate->buffer_1 = NULL;
  }
  if (qstate->gbank != NULL) {
    free(qstate->gbank); qstate->gbank = NULL;
  }

  checkCudaErrors(hipFree(qstate->d_buffer_0)); qstate->d_buffer_0 = NULL;
  checkCudaErrors(hipFree(qstate->d_buffer_1)); qstate->d_buffer_1 = NULL;
  
  free(qstate);
}
  
#ifdef __cplusplus
}
#endif
