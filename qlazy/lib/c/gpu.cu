/*
 *  gpu.cu
 */

#include "qlazy.h"

#ifdef __cplusplus
extern "C" {
#endif

bool gpu_preparation(void)
{
  int *dummy;
  checkCudaErrors(hipMalloc((void**)&dummy,sizeof(int)*1));
  checkCudaErrors(hipFree(dummy));

  SUC_RETURN(true);
}

#ifdef __cplusplus
}
#endif
